#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY,  int maxIterations, int* img, int pitch, float group_size) {
    // To avoid error caused by the floating number, use the following pseudo code
    
    float thisX = (blockIdx.x * blockDim.x + threadIdx.x)*group_size;
    float thisY = (blockIdx.y * blockDim.y + threadIdx.y)*group_size;

    for(float row = 0; row < group_size; row++){
        for(float col = 0; col < group_size; col++){
            
            float x = lowerX + (thisX + col) * stepX;
            float y = lowerY + (thisY + row) * stepY;

            float z_re = x, z_im = y;
            int i;
            for (i = 0; i < maxIterations; ++i)
            {

                if (z_re * z_re + z_im * z_im > 4.f)
                    break;

                float new_re = z_re * z_re - z_im * z_im;
                float new_im = 2.f * z_re * z_im;
                z_re = x + new_re;
                z_im = y + new_im;
            }

            int *rowHead;
            rowHead = (int*)((char*)img + int(thisY + row) * pitch);
            rowHead[int(thisX + col)] = i;
        }
    }
    
    // int idx = int(thisY) * width + int(thisX);
    // img[idx] = i;


}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    hipSetDeviceFlags (hipDeviceMapHost);

    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    size_t size = resX * resY * sizeof(int);
    int *img_h;
    int *img_gpu;
    size_t pitch;

    hipHostAlloc(&img_h, size, hipHostMallocWriteCombined|hipHostMallocMapped);
    // cudaMalloc(&img_gpu, size);
    
    hipHostGetDevicePointer((void**)&img_gpu, (void*)img_h, 0);
    hipMallocPitch((void**)&img_gpu, &pitch, sizeof(int)*resX, resY);
    
    float group_size = 2;
    dim3 threadPerBlock(32,30);
    dim3 numBlocks((resX/int(group_size)) / threadPerBlock.x, (resY/int(group_size)) / threadPerBlock.y);
    mandelKernel <<<numBlocks, threadPerBlock >>> (stepX, stepY, lowerX, lowerY, maxIterations, img_gpu, pitch, group_size);
    hipDeviceSynchronize();

    hipMemcpy2D(img, resX*sizeof(int), img_gpu, pitch, resX*sizeof(int), resY, hipMemcpyHostToHost);
    // for(size_t i = 0; i < resY; ++i){
    //     memcpy(&img[i*resX], &img_gpu[i*pitch], resX * sizeof(int));
    // }

    hipFree(img_gpu);
    hipHostFree(img_h);
}
